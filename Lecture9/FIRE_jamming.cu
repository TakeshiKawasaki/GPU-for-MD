#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <float.h>
#include <stdio.h>
#include <time.h>
#include "../timer.cuh"
#include <math.h>
#include <iostream>
#include <fstream>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include "../MT.h"
using namespace std;

//Using "const", the variable is shared into both gpu and cpu. 
const int  NT = 1024; //Num of the cuda threads.
const int  NP = 2e+3; //Particle number.
const int  NB = (NP+NT-1)/NT; //Num of the cuda blocks.
const int  NN = 100;
const int  NPC = 1000; // Number of the particles in the neighbour cell 
const double dt0 = 0.01;
const double dtmax=0.05;
const double dtmin=0.001;
const double RCHK= 2.0;
const double rcut= 1.0;
const double phi = 0.90;
const double f_thresh= 1.e-12;

//Initialization of "hiprandState"
__global__ void setCurand(unsigned long long seed, hiprandState *state){
  int i_global = threadIdx.x + blockIdx.x*blockDim.x;
  hiprand_init(seed, i_global, 0, &state[i_global]);
}

__global__ void eom_kernel(double*x_dev,double*y_dev,double *vx_dev,double *vy_dev,double *fx_dev,double *fy_dev,double *L_dev,double *dt_dev, int *FIRE_gate_dev){
  int i_global = threadIdx.x + blockIdx.x*blockDim.x;

  if(i_global<NP){
    vx_dev[i_global] +=  fx_dev[i_global]*dt_dev[0];
    vy_dev[i_global] +=  fy_dev[i_global]*dt_dev[0];
    x_dev[i_global]  +=  vx_dev[i_global]*dt_dev[0];
    y_dev[i_global]  +=  vy_dev[i_global]*dt_dev[0];
    x_dev[i_global]  -= (*L_dev)*floor(x_dev[i_global]/(*L_dev));
    y_dev[i_global]  -= (*L_dev)*floor(y_dev[i_global]/(*L_dev));
  }
   if(i_global == 0){
      FIRE_gate_dev[0] = 1;
   }
}

__global__ void FIRE_synth_dev(double *vx_dev,double *vy_dev, double *fx_dev, double *fy_dev, double *power_dev,double *alpha_dev,int *FIRE_gate_dev){
  int i_global = threadIdx.x + blockIdx.x*blockDim.x;
  double f,v;
  if(i_global<NP){
    f = sqrt(fx_dev[i_global]*fx_dev[i_global]+fy_dev[i_global]*fy_dev[i_global]);
    v = sqrt(vx_dev[i_global]*vx_dev[i_global]+vy_dev[i_global]*vy_dev[i_global]);
    vx_dev[i_global] = (1.-alpha_dev[0])*vx_dev[i_global]+alpha_dev[0]*v*fx_dev[i_global]/(f+DBL_EPSILON);
    vy_dev[i_global] = (1.-alpha_dev[0])*vy_dev[i_global]+alpha_dev[0]*v*fy_dev[i_global]/(f+DBL_EPSILON);
    power_dev[i_global] = vx_dev[i_global]*fx_dev[i_global]+vy_dev[i_global]*fy_dev[i_global];
    if(f > f_thresh){
      FIRE_gate_dev[0]=0;      
    }
  }
  //if(i_global==0)
   // printf("f=%.25f,fx=%.25f,fy=%.25f\n",f,fx_dev[0],fy_dev[0]);
}

__global__ void FIRE_reset_dev(double *vx_dev, double *vy_dev,double *power_dev,double *alpha_dev,double *dt_dev,int *FIRE_param_gate_dev){
  int i_global = threadIdx.x + blockIdx.x*blockDim.x;
  
  if(i_global<NP){
    if(power_dev[0] < 0){
      vx_dev[i_global] = 0.0; vy_dev[i_global] = 0.0;
      if(i_global == 0){
	       alpha_dev[0] = 0.1;
	       dt_dev[0] *= 0.5;
         FIRE_param_gate_dev[0]=0;
      }
    }
    else{ //this should be changed into five times criterion
      FIRE_param_gate_dev[0]++;
      if(i_global == 0 && FIRE_param_gate_dev[0]>4){
       //printf("power=%.25f,alpha=%.16f,dt=%f\n",power_dev[0],alpha_dev[0],dt_dev[0]);
	     alpha_dev[0] *= 0.99;
       if(dt_dev[0] < dtmax)
   	     dt_dev[0] *= 1.1;
        FIRE_param_gate_dev[0]=0;
      }
    }
  }
}

__global__ void disp_gate_kernel(double *vx_dev,double *vy_dev,double *dx_dev,double *dy_dev,int *gate_dev,double *dt_dev)
{
  double r2;  
  int i_global = threadIdx.x + blockIdx.x*blockDim.x;
  
  if(i_global<NP){
    dx_dev[i_global]+=vx_dev[i_global]*dt_dev[0];
    dy_dev[i_global]+=vy_dev[i_global]*dt_dev[0];
    r2 = dx_dev[i_global]*dx_dev[i_global]+dy_dev[i_global]*dy_dev[i_global];
    if(r2> 0.25*(RCHK-rcut)*(RCHK-rcut)){
      gate_dev[0]=1;
    }
  }
}

__global__ void update(double *L_dev,double *x_dev,double *y_dev,double *dx_dev,double *dy_dev,int *list_dev,int *gate_dev)
{
  double dx,dy,r2;  
  int i_global = threadIdx.x + blockIdx.x*blockDim.x;
  
  if(gate_dev[0] == 1 && i_global<NP){
    
    list_dev[NN*i_global]=0;      
    for (int j=0; j<NP; j++)
      if(j != i_global){
      	dx = x_dev[i_global] - x_dev[j];
	      dy = y_dev[i_global] - y_dev[j];
	      dx -= (*L_dev)*floor(dx/(*L_dev)+0.5);
	      dy -= (*L_dev)*floor(dy/(*L_dev)+0.5);	  
	      r2 = dx*dx + dy*dy;
	      if(r2 < RCHK*RCHK){
	        list_dev[NN*i_global]++;
	        list_dev[NN*i_global+list_dev[NN*i_global]]=j;
	      }
      }
    //    printf("i=%d, list=%d\n",i_global,list_dev[NN*i_global]);      
    dx_dev[i_global]=0.;
    dy_dev[i_global]=0.;
  }
}

__device__ int f(int i,int M){
  int k;
  k=i;
  if(k>=M)
    k-=M;
  if(k<0)
    k+=M;
  return k;
}

__global__ void cell_map(double *L_dev,double *x_dev,double *y_dev,int *map_dev,int *gate_dev, int M)
{  
  int i_global = threadIdx.x + blockIdx.x*blockDim.x;
  int nx,ny;
  int num;
  if(gate_dev[0] == 1 && i_global<NP){
    nx = f((int)(x_dev[i_global]*(double)M/(double)(*L_dev)),M);
    ny = f((int)(y_dev[i_global]*(double)M/(double)(*L_dev)),M);
    num = atomicAdd(&map_dev[(nx+M*ny)*NPC],1);
    map_dev[(nx+M*ny)*NPC+num+1] = i_global;
  }
}
  
__global__ void cell_list(double *L_dev,double *x_dev,double *y_dev,double *dx_dev,double *dy_dev,int *list_dev,int *map_dev,int *gate_dev, int M)
{
  int i_global = threadIdx.x + blockIdx.x*blockDim.x;
  int nx,ny;
  int j,k;
  double dx,dy,r2;  
  int l,m;
  if(gate_dev[0] == 1 && i_global<NP){
    list_dev[NN*i_global]=0;
    nx=f((int)(x_dev[i_global]*(double)M/(double)(*L_dev)),M);
    ny=f((int)(y_dev[i_global]*(double)M/(double)(*L_dev)),M);
    for(m=ny-1;m<=ny+1;m++)
      for(l=nx-1;l<=nx+1;l++){
	        for(k=1; k<=map_dev[(f(l,M)+M*f(m,M))*NPC]; k++){
	          j = map_dev[(f(l,M)+M*f(m,M))*NPC+k];
	          if(j != i_global){
	            dx =x_dev[i_global] - x_dev[j];
	            dy =y_dev[i_global] - y_dev[j];
	            dx -=(*L_dev)*floor(dx/(*L_dev)+0.5);
	            dy -=(*L_dev)*floor(dy/(*L_dev)+0.5);	  
	            r2 = dx*dx + dy*dy;
	            if(r2 < RCHK*RCHK){
	              list_dev[NN*i_global]++;
	              list_dev[NN*i_global+list_dev[NN*i_global]]=j;
	             // printf("i=%d, list=%d\n",i_global,list_dev[NN*i_global]);     
	            }
	          }
	       }
    }
    //    printf("i=%d, list=%d\n",i_global,list_dev[NN*i_global]);      
    dx_dev[i_global]=0.;
    dy_dev[i_global]=0.;
  } 
}

__global__ void calc_force_kernel(double*x_dev,double*y_dev,double *fx_dev,double *fy_dev,double *a_dev,double *L_dev,int *list_dev){
  double dx,dy,dr,dU_r,a_ij;
  int i_global = threadIdx.x + blockIdx.x*blockDim.x;
  if(i_global<NP){
    fx_dev[i_global]=0.0;
    fy_dev[i_global]=0.0;
    for(int j = 1; j<=list_dev[NN*i_global]; j++){
      dx = x_dev[list_dev[NN*i_global+j]]-x_dev[i_global];
      dy = y_dev[list_dev[NN*i_global+j]]-y_dev[i_global];
      dx -= (*L_dev)*floor(dx/(*L_dev)+0.5);
      dy -= (*L_dev)*floor(dy/(*L_dev)+0.5);	
      dr = sqrt(dx*dx+dy*dy);
      a_ij = 0.5*(a_dev[i_global]+a_dev[list_dev[NN*i_global+j]]);
      if(dr < a_ij){
	      dU_r = -(1-dr/a_ij)/a_ij; //derivertive of U wrt r.
	      fx_dev[i_global] += dU_r*dx/dr;
	      fy_dev[i_global] += dU_r*dy/dr;
      }
    }
    // printf("i=%d, fx=%f\n",i_global,fx_dev[i_global]);
  }
}


__global__ void calc_energy_kernel(double*x_dev,double*y_dev,double *pot_dev,double *a_dev,double *L_dev,int *list_dev){
  double dx,dy,dr,a_ij;
  int i_global = threadIdx.x + blockIdx.x*blockDim.x;
  if(i_global<NP){
    pot_dev[i_global]=0.0;
    for(int j = 1; j<=list_dev[NN*i_global]; j++){
      dx = x_dev[list_dev[NN*i_global+j]]-x_dev[i_global];
      dy = y_dev[list_dev[NN*i_global+j]]-y_dev[i_global];
      dx -= (*L_dev)*floor(dx/(*L_dev)+0.5);
      dy -= (*L_dev)*floor(dy/(*L_dev)+0.5);
      dr = sqrt(dx*dx+dy*dy);
      a_ij= 0.5*(a_dev[i_global]+a_dev[list_dev[NN*i_global+j]]);
      if(dr < a_ij)
	      pot_dev[i_global]+= 0.5*(1.-dr/a_ij)*(1.-dr/a_ij);
    }
  }
}

__global__ void copy_kernel(double *x0_dev, double *y0_dev, double *x_dev, double *y_dev){
  int i_global = threadIdx.x + blockIdx.x*blockDim.x;
  x0_dev[i_global]=x_dev[i_global];
  y0_dev[i_global]=y_dev[i_global];
}

__global__ void init_gate_kernel(int *gate_dev, int c){
  gate_dev[0]=c;
}
__global__ void init_scalar_kernel(double *a_dev, double d){
  a_dev[0]=d;
}


__global__ void init_map_kernel(int *map_dev,int M){
  int i_global = threadIdx.x + blockIdx.x*blockDim.x;
  map_dev[i_global] = 0;
}

__global__ void init_array(double *x_dev, double c){
  int i_global = threadIdx.x + blockIdx.x*blockDim.x;
  x_dev[i_global] = c;
}

__global__ void init_diamters(double *a_dev){
  int i_global = threadIdx.x + blockIdx.x*blockDim.x;
  if(i_global<NP/2)
    a_dev[i_global] = 1.0;
  if(i_global>NP/2 && i_global<NP)
    a_dev[i_global] = 1.4;
}

__global__ void init_array_rand(double *x_dev, double c,hiprandState *state){
  int i_global = threadIdx.x + blockIdx.x*blockDim.x;
  x_dev[i_global] = c*hiprand_uniform(&state[i_global]);
}

__global__ void volume_affine(double *x_dev, double *y_dev,double *phi_dev,double *deltaphi_dev,double *L_dev){
  int i_global = threadIdx.x + blockIdx.x*blockDim.x;
  if(i_global<NP){
    x_dev[i_global] = x_dev[i_global]*sqrt(*phi_dev/(*phi_dev+*deltaphi_dev));
    y_dev[i_global] = y_dev[i_global]*sqrt(*phi_dev/(*phi_dev+*deltaphi_dev));
  }
  if(i_global==0){
    *phi_dev += *deltaphi_dev;
    *L_dev   *= sqrt(*phi_dev/(*phi_dev+*deltaphi_dev));
  }
}

__global__ void add_reduction(double *pot_dev, int *reduce_dev, int *remain_dev){
  int i_global = threadIdx.x + blockIdx.x*blockDim.x;
  if(i_global< *reduce_dev)
    pot_dev[i_global] += pot_dev[i_global+*remain_dev];
}

__global__ void len_ini(int *reduce_dev,int *remain_dev, int size){
  int i_global = threadIdx.x + blockIdx.x*blockDim.x;
  if(i_global==0){
    *reduce_dev= size/2;
    *remain_dev= size - *reduce_dev; 
  }
}
__global__ void len_div(int *reduce_dev,int *remain_dev){
  int i_global = threadIdx.x + blockIdx.x*blockDim.x;
  if(i_global==0){
    *reduce_dev = *remain_dev/2;
    *remain_dev -= *reduce_dev;
  }
}

int main(){
  double *x,*vx,*y,*vy,*fx,*fy,*pot,*a,*x_dev,*vx_dev,*y_dev,*dx_dev,*dy_dev,*vy_dev,*pot_dev,*a_dev,*fx_dev,*fy_dev,*power_dev,*L_dev;
  double *dt_dev,*alpha_dev,*phi_dev;
  int *list_dev,*map_dev,*gate_dev,*remain_dev,*reduce_dev;
  int *FIRE_gate_dev,FIRE_gate,*FIRE_param_gate_dev;
  int clock=0;
  hiprandState *state; //Cuda state for random numbers
  double sec; //measurred time
  double L = sqrt(M_PI*(1.0*1.0+1.4*1.4)*(double)NP/(8.*phi));
  int M = (int)(L/RCHK);
  cout <<"M="<< M <<"L="<<L<<endl;

  x  = (double*)malloc(NB*NT*sizeof(double));
  y  = (double*)malloc(NB*NT*sizeof(double));
  vx = (double*)malloc(NB*NT*sizeof(double));
  vy = (double*)malloc(NB*NT*sizeof(double));
  a  = (double*)malloc(NB*NT*sizeof(double));
  fx  = (double*)malloc(NB*NT*sizeof(double));
  fy  = (double*)malloc(NB*NT*sizeof(double));
  pot  = (double*)malloc(NB*NT*sizeof(double));
  hipMalloc((void**)&x_dev,  NB * NT * sizeof(double)); // CudaMalloc should be executed once
  hipMalloc((void**)&y_dev,  NB * NT * sizeof(double)); 
  hipMalloc((void**)&dx_dev,  NB * NT * sizeof(double)); 
  hipMalloc((void**)&dy_dev,  NB * NT * sizeof(double)); 
  hipMalloc((void**)&vx_dev, NB * NT * sizeof(double)); 
  hipMalloc((void**)&vy_dev, NB * NT * sizeof(double));
  hipMalloc((void**)&fx_dev, NB * NT * sizeof(double)); 
  hipMalloc((void**)&fy_dev, NB * NT * sizeof(double));
  hipMalloc((void**)&pot_dev, NB * NT * sizeof(double));
  hipMalloc((void**)&a_dev,  NB * NT * sizeof(double)); 
  hipMalloc((void**)&power_dev,  NB * NT * sizeof(double));
  hipMalloc((void**)&dt_dev, sizeof(double)); 
  hipMalloc((void**)&alpha_dev, sizeof(double));
  hipMalloc((void**)&L_dev, sizeof(double)); 
  hipMalloc((void**)&phi_dev, sizeof(double)); 
  hipMalloc((void**)&gate_dev, sizeof(int));
  hipMalloc((void**)&FIRE_gate_dev, sizeof(int)); 
   hipMalloc((void**)&FIRE_param_gate_dev, sizeof(int)); 
  hipMalloc((void**)&remain_dev, sizeof(int));
  hipMalloc((void**)&reduce_dev, sizeof(int));
  hipMalloc((void**)&list_dev,  NB * NT * NN* sizeof(int)); 
  hipMalloc((void**)&map_dev,  M * M * NPC* sizeof(int)); 
  hipMalloc((void**)&state,  NB * NT * sizeof(hiprandState)); 
  hipMemcpy(L_dev, &L,sizeof(double),hipMemcpyHostToDevice);
  hipMemcpy(phi_dev,&phi,sizeof(double),hipMemcpyHostToDevice);
  setCurand<<<NB,NT>>>(0, state); // Construction of the cudarand state.  
  init_array_rand<<<NB,NT>>>(x_dev,L,state);
  init_array_rand<<<NB,NT>>>(y_dev,L,state);
  init_diamters<<<NB,NT>>>(a_dev);
  init_array<<<NB,NT>>>(vx_dev,0.);
  init_array<<<NB,NT>>>(vy_dev,0.);
  init_array<<<NB,NT>>>(pot_dev,0.);
  init_gate_kernel<<<1,1>>>(gate_dev,1);
  init_gate_kernel<<<1,1>>>(FIRE_gate_dev,0);
  init_gate_kernel<<<1,1>>>(FIRE_param_gate_dev,0);
  init_scalar_kernel<<<1,1>>>(dt_dev,dt0);
  init_scalar_kernel<<<1,1>>>(alpha_dev,0.1);
  init_map_kernel<<<M*M,NPC>>>(map_dev,M);
  cell_map<<<NB,NT>>>(L_dev,x_dev,y_dev,map_dev,gate_dev,M);
  cell_list<<<NB,NT>>>(L_dev,x_dev,y_dev,dx_dev,dy_dev,list_dev,map_dev,gate_dev,M);
 
  measureTime();  
  for(;;){
    clock++;
    calc_force_kernel<<<NB,NT>>>(x_dev,y_dev,fx_dev,fy_dev,a_dev,L_dev,list_dev);
    eom_kernel<<<NB,NT>>>(x_dev,y_dev,vx_dev,vy_dev,fx_dev,fy_dev,L_dev,dt_dev,FIRE_gate_dev);
    FIRE_synth_dev<<<NB,NT>>>(vx_dev,vy_dev,fx_dev,fy_dev,power_dev,alpha_dev,FIRE_gate_dev);
    len_ini<<<1,1>>>(reduce_dev,remain_dev,NP);
    int reduce=NP/2,remain=NP-NP/2;
    while(reduce>0){
      add_reduction<<<(reduce+NT-1)/NT,NT>>>(power_dev,reduce_dev,remain_dev);
      reduce = remain/2;remain-=reduce;
      len_div<<<1,1>>>(reduce_dev,remain_dev);
    }
    FIRE_reset_dev<<<NB,NT>>>(vx_dev,vy_dev,power_dev,alpha_dev,dt_dev,FIRE_param_gate_dev);
    init_gate_kernel<<<1,1>>>(gate_dev,0);
    disp_gate_kernel<<<NB,NT>>>(vx_dev,vy_dev,dx_dev,dy_dev,gate_dev,dt_dev);
    init_map_kernel<<<M*M,NPC>>>(map_dev,M);
    cell_map<<<NB,NT>>>(L_dev,x_dev,y_dev,map_dev,gate_dev,M);
    cell_list<<<NB,NT>>>(L_dev,x_dev,y_dev,dx_dev,dy_dev,list_dev,map_dev,gate_dev,M);
  //////////////////////////
   // if(clock%1000==0)
      hipMemcpy(&FIRE_gate,FIRE_gate_dev,sizeof(int),hipMemcpyDeviceToHost);
  //  cout<<FIRE_gate<<endl;
    if(FIRE_gate == 1){
      hipMemcpy(fx,fx_dev, NB*NT*sizeof(double),hipMemcpyDeviceToHost);
      hipMemcpy(fy,fy_dev, NB*NT*sizeof(double),hipMemcpyDeviceToHost);
      cout<<"count= "<< clock <<" fx= "<<fx[0]<<" fy= "<<fy[0]<<endl;
      break;
    }
  //////////////////////////
  }
  sec = measureTime()/1000.;
  cout<<"time(sec):"<<sec<<endl;
  hipFree(x_dev);
  hipFree(vx_dev);
  hipFree(y_dev);
  hipFree(vy_dev);
  hipFree(dx_dev);
  hipFree(dy_dev);
  hipFree(pot_dev);
  hipFree(gate_dev);
  hipFree(dt_dev);
  hipFree(alpha_dev);
  hipFree(L_dev);
  hipFree(state);
  free(x); 
  free(vx); 
  free(y); 
  free(vy);
  free(pot); 
  return 0;
}
