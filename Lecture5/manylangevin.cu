#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#include "../timer.cuh"
#include <math.h>
#include <iostream>
#include <fstream>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include "../MT.h"
using namespace std;

//Using "const", the variable is shared into both gpu and cpu.
const int  NT = 1024; //Num of the cuda threads.
const int  NP = 1e+4; //Particle number.
const int  NB = (NP+NT-1)/NT; //Num of the cuda blocks.
const double dt = 0.01;
const int timemax = 1.e+2;
//Langevin parameters
const double zeta = 1.0;
const double temp = 1.e-4;
const double rho = 0.85;
//const double LB = 100.;


//Initiallization of "hiprandState"
__global__ void setCurand(unsigned long long seed, hiprandState *state){
  int i_global = threadIdx.x + blockIdx.x*blockDim.x;
  hiprand_init(seed, i_global, 0, &state[i_global]);
}


//Gaussian random number's generation
__global__ void langevin_kernel(double*x_dev,double*y_dev,double *vx_dev,double *vy_dev,double *fx_dev,double *fy_dev,hiprandState *state, double noise_intensity,double LB){
  int i_global = threadIdx.x + blockIdx.x*blockDim.x;
  if(i_global<NP){
    //  printf("%d,%f\n",i_global,v_dev[i_global]);
    vx_dev[i_global] += -zeta*vx_dev[i_global]*dt+ fx_dev[i_global]*dt + noise_intensity*hiprand_normal(&state[i_global]);
    vy_dev[i_global] += -zeta*vy_dev[i_global]*dt+ fy_dev[i_global]*dt + noise_intensity*hiprand_normal(&state[i_global]);
    x_dev[i_global] += vx_dev[i_global]*dt;
    y_dev[i_global] += vy_dev[i_global]*dt;

    x_dev[i_global]  -= LB*floor(x_dev[i_global]/LB);
    y_dev[i_global]  -= LB*floor(y_dev[i_global]/LB);
  //  printf("vx=%f\n",vx_dev[i_global]);
  }

}

//Force calculation NP*NP matrix...
__global__ void calc_force_kernel(double*x_dev,double*y_dev,double *fx_dev,double *fy_dev,double *a_dev,double LB){
  double dx,dy,dr,dU,a_ij;
  int i_global = threadIdx.x + blockIdx.x*blockDim.x;



  if(i_global<NP){
  fx_dev[i_global]=0.0;
  fy_dev[i_global]=0.0;
  for(int j = 0; j<NP; j++)
      if(j != i_global){
	dx=x_dev[j]-x_dev[i_global];
	dy=y_dev[j]-y_dev[i_global];

	dx -= LB*floor(dx/LB+0.5);
	dy -= LB*floor(dy/LB+0.5);

	dr = sqrt(dx*dx+dy*dy);
	a_ij=0.5*(a_dev[i_global]+a_dev[j]);

	if(dr < a_ij){
	  dU = -(1-dr/a_ij)/a_ij; //derivertive of U wrt r.
	  fx_dev[i_global] += dU*dx/dr;
	  fy_dev[i_global] += dU*dy/dr;
	}
      }
  // printf("i=%d, fx=%f\n",i_global,fx_dev[i_global]);
  }
}

__global__ void init_array_rand(double *x_dev, double c,hiprandState *state){
  int i_global = threadIdx.x + blockIdx.x*blockDim.x;
  x_dev[i_global] = c*hiprand_uniform(&state[i_global]);
}

void init_array(double *x,int Narr, double c){
  for(int i=0;i<Narr;i++)
    x[i] = c;
}

//void init_array_rand(double *x,int Narr, double c){
 // for(int i=0;i<Narr;i++)
   // x[i] = c*rand()/RAND_MAX;
//}

void output(double *x,double *y,double *vx,double *vy,double *a){
  static int count=1;
  char filename[128];
  sprintf(filename,"coord_%.d.dat",count);
  ofstream file;
  file.open(filename);
  double kine=0.0;

  for(int i=0;i<NP;i++){
    file << x[i] << " " << y[i]<< " " << a[i] << endl;
  //  cout << vx[i] << endl;

    kine+=0.5*(vx[i]*vx[i]+vy[i]*vy[i]);
  }
  file.close();
  cout<<"temp="<< kine/(double)NP <<endl;
  count++;
}


int main(){
  double *x,*vx,*y,*vy,*a,*x_dev,*vx_dev,*y_dev,*vy_dev,*a_dev,*fx_dev,*fy_dev;
  hiprandState *state; //Cuda state for random numbers
  double sec; //measurred time
  double noise_intensity = sqrt(2.*zeta*temp*dt); //Langevin noise intensity.
  double LB = sqrt(M_PI*1.0*1.0*(double)NP*0.25/rho);
  x  = (double*)malloc(NB*NT*sizeof(double));
  y  = (double*)malloc(NB*NT*sizeof(double));
  vx = (double*)malloc(NB*NT*sizeof(double));
  vy = (double*)malloc(NB*NT*sizeof(double));
  a  = (double*)malloc(NB*NT*sizeof(double));
  hipMalloc((void**)&x_dev,  NB * NT * sizeof(double)); // CudaMalloc should be executed once in the host.
  hipMalloc((void**)&y_dev,  NB * NT * sizeof(double));
  hipMalloc((void**)&vx_dev, NB * NT * sizeof(double));
  hipMalloc((void**)&vy_dev, NB * NT * sizeof(double));
  hipMalloc((void**)&fx_dev, NB * NT * sizeof(double));
  hipMalloc((void**)&fy_dev, NB * NT * sizeof(double));
  hipMalloc((void**)&a_dev,  NB * NT * sizeof(double));
  hipMalloc((void**)&state,  NB * NT * sizeof(hiprandState));

 // init_array_rand(x,NT*NB,LB);
 // init_array_rand(y,NT*NB,LB);
  init_array(a,NT*NB,1.0);
  init_array(vx,NT*NB,0.);
  init_array(vy,NT*NB,0.);

  hipMemcpy(x_dev,   x, NB * NT* sizeof(double),hipMemcpyHostToDevice);
  hipMemcpy(y_dev,   y, NB * NT* sizeof(double),hipMemcpyHostToDevice);
  hipMemcpy(vx_dev, vx, NB * NT* sizeof(double),hipMemcpyHostToDevice);
  hipMemcpy(vy_dev, vy, NB * NT* sizeof(double),hipMemcpyHostToDevice);
  hipMemcpy(a_dev,  a, NB * NT* sizeof(double),hipMemcpyHostToDevice);

  setCurand<<<NB,NT>>>(0, state); // Construction of the cudarand state.
  init_array_rand<<<NB,NT>>>(x_dev,LB,state);
  init_array_rand<<<NB,NT>>>(y_dev,LB,state);

  for(double t=0;t<timemax;t+=dt){
    calc_force_kernel<<<NB,NT>>>(x_dev,y_dev,fx_dev,fy_dev,a_dev,LB);
    langevin_kernel<<<NB,NT>>>(x_dev,y_dev,vx_dev,vy_dev,fx_dev,fy_dev,state,noise_intensity,LB);
    //     hipDeviceSynchronize(); // for printf in the device.
  }
  hipMemcpy(x,   x_dev, NB * NT* sizeof(double),hipMemcpyDeviceToHost);
  hipMemcpy(vx, vx_dev, NB * NT* sizeof(double),hipMemcpyDeviceToHost);
  hipMemcpy(y,   y_dev, NB * NT* sizeof(double),hipMemcpyDeviceToHost);
  hipMemcpy(vy, vy_dev, NB * NT* sizeof(double),hipMemcpyDeviceToHost);

  // for(int i=0;i<NP;i++)
  //  cout<<i<<" "<<vx[i]<<endl;

  output(x,y,vx,vy,a);

  hipFree(x_dev);
  hipFree(vx_dev);
  hipFree(y_dev);
  hipFree(vy_dev);
  hipFree(state);
  free(x);
  free(vx);
  free(y);
  free(vy);
  return 0;
}
