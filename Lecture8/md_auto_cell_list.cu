#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#include "../timer.cuh"
#include <math.h>
#include <iostream>
#include <fstream>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include "../MT.h"
using namespace std;

//Using "const", the variable is shared into both gpu and cpu. 
const int  NT = 1024; //Num of the cuda threads.
const int  NP = 1e+4; //Particle number.
const int  NB = (NP+NT-1)/NT; //Num of the cuda blocks.
const int  NN = 100;
const int  NPC = 1000; // Number of the particles in the neighbour cell 
const double dt_bd = 0.01;
const double dt_md = 0.002;
const int timemax_bd = 3e+4;
const int timemax_md = 0.5e+4;
//Langevin parameters
const double zeta = 1.0;
const double temp = 1.e-4;
const double rho = 0.85;
const double RCHK= 2.0;
const double rcut= 1.0;


//Initiallization of "hiprandState"
__global__ void setCurand(unsigned long long seed, hiprandState *state){
  int i_global = threadIdx.x + blockIdx.x*blockDim.x;
  hiprand_init(seed, i_global, 0, &state[i_global]);
}

//Gaussian random number's generation
__global__ void genrand_kernel(double *result, hiprandState *state, double cons){  
  int i_global = threadIdx.x + blockIdx.x*blockDim.x;
  result[i_global] = cons*hiprand_normal(&state[i_global]);
}

//Gaussian random number's generation
__global__ void langevin_kernel(double*x_dev,double*y_dev,double *vx_dev,double *vy_dev,double *fx_dev,double *fy_dev,hiprandState *state, double noise_intensity,double LB){
  int i_global = threadIdx.x + blockIdx.x*blockDim.x;

  if(i_global<NP){
    vx_dev[i_global] += -zeta*vx_dev[i_global]*dt_bd+ fx_dev[i_global]*dt_bd + noise_intensity*hiprand_normal(&state[i_global]);
    vy_dev[i_global] += -zeta*vy_dev[i_global]*dt_bd+ fy_dev[i_global]*dt_bd + noise_intensity*hiprand_normal(&state[i_global]);
    x_dev[i_global] += vx_dev[i_global]*dt_bd;
    y_dev[i_global] += vy_dev[i_global]*dt_bd;
    x_dev[i_global]  -= LB*floor(x_dev[i_global]/LB);
    y_dev[i_global]  -= LB*floor(y_dev[i_global]/LB);
  }
}


__global__ void md_kernel_pre(double*x_dev,double*y_dev,double *vx_dev,double *vy_dev,double *fx_dev,double *fy_dev,double LB){
  int i_global = threadIdx.x + blockIdx.x*blockDim.x;

  if(i_global<NP){
    x_dev[i_global] += vx_dev[i_global]*dt_md + 0.5*fx_dev[i_global]*dt_md*dt_md;
    y_dev[i_global] += vy_dev[i_global]*dt_md + 0.5*fy_dev[i_global]*dt_md*dt_md;
    vx_dev[i_global] += 0.5*fx_dev[i_global]*dt_md;
    vy_dev[i_global] += 0.5*fy_dev[i_global]*dt_md;
    }
}

__global__ void md_kernel_post(double*x_dev,double*y_dev,double *vx_dev,double *vy_dev,double *fx_dev,double *fy_dev,double LB){
  int i_global = threadIdx.x + blockIdx.x*blockDim.x;

  if(i_global<NP){
    vx_dev[i_global] += 0.5*fx_dev[i_global]*dt_md;
    vy_dev[i_global] += 0.5*fy_dev[i_global]*dt_md;
  }
}


__global__ void disp_gate_kernel(double LB,double *vx_dev,double *vy_dev,double *dx_dev,double *dy_dev,int *gate_dev,double dt)
{
  double r2;  
  int i_global = threadIdx.x + blockIdx.x*blockDim.x;
  
  if(i_global<NP){
    dx_dev[i_global]+=vx_dev[i_global]*dt;
    dy_dev[i_global]+=vy_dev[i_global]*dt;
    r2 = dx_dev[i_global]*dx_dev[i_global]+dy_dev[i_global]*dy_dev[i_global];
    if(r2> 0.25*(RCHK-rcut)*(RCHK-rcut)){
      gate_dev[0]=1;
    }
  }
}


__global__ void update(double LB,double *x_dev,double *y_dev,double *dx_dev,double *dy_dev,int *list_dev,int *gate_dev)
{
  double dx,dy,r2;  
  int i_global = threadIdx.x + blockIdx.x*blockDim.x;
  
  if(gate_dev[0] == 1 && i_global<NP){
    
    list_dev[NN*i_global]=0;      
    for (int j=0; j<NP; j++)
      if(j != i_global){
	dx =x_dev[i_global] - x_dev[j];
	dy =y_dev[i_global] - y_dev[j];
	dx -=LB*floor(dx/LB+0.5);
	dy -=LB*floor(dy/LB+0.5);	  
	r2 = dx*dx + dy*dy;
	if(r2 < RCHK*RCHK){
	  list_dev[NN*i_global]++;
	  list_dev[NN*i_global+list_dev[NN*i_global]]=j;
	}
      }
    //    printf("i=%d, list=%d\n",i_global,list_dev[NN*i_global]);      
    dx_dev[i_global]=0.;
    dy_dev[i_global]=0.;
  }
}

__device__ int f(int i,int M){
  int k;
  k=i;
  if(k>=M)
    k-=M;
  if(k<0)
    k+=M;
  return k;
}



__global__ void cell_map(double LB,double *x_dev,double *y_dev,int *map_dev,int *gate_dev, int M)
{
  
  int i_global = threadIdx.x + blockIdx.x*blockDim.x;
  int nx,ny;
   int num;
  
  if(gate_dev[0] == 1 && i_global<NP){
    
    nx=f((int)(x_dev[i_global]*(double)M/(double)LB),M);
    ny=f((int)(y_dev[i_global]*(double)M/(double)LB),M);
    
    //  for(int m=ny-1;m<=ny+1;m++)
    //  for(int l=nx-1;l<=nx+1;l++){
    num = atomicAdd(&map_dev[(nx+M*ny)*NPC],1);
    // num = map_dev[(nx+M*ny)*NPC]+1;
    // if(num == 0)
    // printf("%d = %d\n",num,map_dev[(nx+M*ny)*NPC]);
    map_dev[(nx+M*ny)*NPC+num+1] = i_global;
    
    //	if(num>70)
    //	printf("i=%d, map_dev=%d, f=%d, MM=%d, num=%d\n",i_global,map_dev[(f(l,M)+M*f(m,M))*NPC + num], f(l,M)+M*f(m,M),M*M,num);
    // }
    //  printf("i=%d\n",i_global);    
    // }
    //  printf("i=%d, map_dev=%d, f=%d, MM=%d, num=%d\n",i_global,map_dev[(f(l,M)+M*f(m,M))*NPC + num], f(l,M)+M*f(m,M),M*M,num);
  }
}
  
  
__global__ void cell_list(double LB,double *x_dev,double *y_dev,double *dx_dev,double *dy_dev,int *list_dev,int *map_dev,int *gate_dev, int M)
{
  int i_global = threadIdx.x + blockIdx.x*blockDim.x;
  int nx,ny;
  int j,k;
  double dx,dy,r2;  
  int l,m;
  //  printf("i=%d \n",i_global); 
  if(gate_dev[0] == 1 && i_global<NP){
    // if(i_global==0)
    // printf("update\n");
    list_dev[NN*i_global]=0;
    
    nx=f((int)(x_dev[i_global]*(double)M/(double)LB),M);
    ny=f((int)(y_dev[i_global]*(double)M/(double)LB),M);
    
    for(m=ny-1;m<=ny+1;m++)
      for(l=nx-1;l<=nx+1;l++){
	
	for(k=1; k<=map_dev[(f(l,M)+M*f(m,M))*NPC]; k++){
	  j = map_dev[(f(l,M)+M*f(m,M))*NPC+k];
	  if(j != i_global){
	    dx =x_dev[i_global] - x_dev[j];
	    dy =y_dev[i_global] - y_dev[j];
	    dx -=LB*floor(dx/LB+0.5);
	    dy -=LB*floor(dy/LB+0.5);	  
	    r2 = dx*dx + dy*dy;
	    if(r2 < RCHK*RCHK){
	      list_dev[NN*i_global]++;
	      list_dev[NN*i_global+list_dev[NN*i_global]]=j;
	      // printf("i=%d, list=%d\n",i_global,list_dev[NN*i_global]);     
	    }
	  }
	}
      }
    //    printf("i=%d, list=%d\n",i_global,list_dev[NN*i_global]);      
    dx_dev[i_global]=0.;
    dy_dev[i_global]=0.;
  } 
}


__global__ void calc_force_kernel(double*x_dev,double*y_dev,double *fx_dev,double *fy_dev,double *a_dev,double LB,int *list_dev){
  double dx,dy,dr,dU_r,a_ij;
  int i_global = threadIdx.x + blockIdx.x*blockDim.x;
 
  
  if(i_global<NP){
    fx_dev[i_global]=0.0;
    fy_dev[i_global]=0.0;

    for(int j = 1; j<=list_dev[NN*i_global]; j++){
      dx=x_dev[list_dev[NN*i_global+j]]-x_dev[i_global];
      dy=y_dev[list_dev[NN*i_global+j]]-y_dev[i_global];
      
      dx -= LB*floor(dx/LB+0.5);
      dy -= LB*floor(dy/LB+0.5);	
      dr = sqrt(dx*dx+dy*dy);
      a_ij= 0.5*(a_dev[i_global]+a_dev[list_dev[NN*i_global+j]]);
      if(dr < a_ij){
	    dU_r = -(1-dr/a_ij)/a_ij; //derivertive of U wrt r.
	    fx_dev[i_global] += dU_r*dx/dr;
	    fy_dev[i_global] += dU_r*dy/dr;
      }
    }
    // printf("i=%d, fx=%f\n",i_global,fx_dev[i_global]);
  }
}


__global__ void calc_energy_kernel(double*x_dev,double*y_dev,double *pot_dev,double *a_dev,double LB,int *list_dev){
  double dx,dy,dr,a_ij;
  int i_global = threadIdx.x + blockIdx.x*blockDim.x;

  if(i_global<NP){
    pot_dev[i_global]=0.0;
    for(int j = 1; j<=list_dev[NN*i_global]; j++){
      dx=x_dev[list_dev[NN*i_global+j]]-x_dev[i_global];
      dy=y_dev[list_dev[NN*i_global+j]]-y_dev[i_global];
      dx -= LB*floor(dx/LB+0.5);
      dy -= LB*floor(dy/LB+0.5);
      dr = sqrt(dx*dx+dy*dy);
      a_ij= 0.5*(a_dev[i_global]+a_dev[list_dev[NN*i_global+j]]);
      if(dr < a_ij)
	pot_dev[i_global]+= 0.5*(1.-dr/a_ij)*(1.-dr/a_ij);
    }
  }
}

__global__ void copy_kernel(double *x0_dev, double *y0_dev, double *x_dev, double *y_dev){
  int i_global = threadIdx.x + blockIdx.x*blockDim.x;
  x0_dev[i_global]=x_dev[i_global];
  y0_dev[i_global]=y_dev[i_global];
  // printf("%f,%f\n",x_dev[i_global],x0_dev[i_global]);
}

__global__ void init_gate_kernel(int *gate_dev, int c){
  gate_dev[0]=c;
}

__global__ void init_map_kernel(int *map_dev,int M){
  int i_global = threadIdx.x + blockIdx.x*blockDim.x;
  // for(int i=0;i<M;i++)
  //  for(int j=0;j<M;j++)
  // map_dev[(i+M*j)*NPC] = 0;
  map_dev[i_global] = 0;
}

__global__ void init_array(double *x_dev, double c){
  int i_global = threadIdx.x + blockIdx.x*blockDim.x;
  x_dev[i_global] = c;
}

__global__ void init_array_rand(double *x_dev, double c,hiprandState *state){
  int i_global = threadIdx.x + blockIdx.x*blockDim.x;
  x_dev[i_global] = c*hiprand_uniform(&state[i_global]);
}

void output(double *x,double *y,double *vx,double *vy,double *a,double *pot){
  static int count=1;
  char filename[128];
  sprintf(filename,"coord_%.d.dat",count);
  ofstream file;
  file.open(filename);
  double temp_ave=0.0;
  double pot_ave=0.0;
  
  for(int i=0;i<NP;i++){
    file << x[i] << " " << y[i]<< " " << a[i] << endl;
    temp_ave+= 0.5*(vx[i]*vx[i]+vy[i]*vy[i]);
    pot_ave += 0.5*pot[i];
  }
  file.close();
  cout<<"temp="<< temp_ave/NP <<" pot= "<<pot_ave/NP<< " tot=" <<(temp_ave + pot_ave)/NP <<endl;
  count++;
}


int main(){
  double *x,*vx,*y,*vy,*a,*pot,*x_dev,*vx_dev,*y_dev,*dx_dev,*dy_dev,*vy_dev,*a_dev,*fx_dev,*fy_dev,*pot_dev;
  int *list_dev,*map_dev,*gate_dev;
  hiprandState *state; //Cuda state for random numbers
  double sec; //measurred time
  double noise_intensity = sqrt(2.*zeta*temp*dt_bd); //Langevin noise intensity.   
  double LB = sqrt(M_PI*1.0*1.0*(double)NP*0.25/rho);
  int M = (int)(LB/RCHK);
  cout <<M<<endl;

  x  = (double*)malloc(NB*NT*sizeof(double));
  y  = (double*)malloc(NB*NT*sizeof(double));
  vx = (double*)malloc(NB*NT*sizeof(double));
  vy = (double*)malloc(NB*NT*sizeof(double));
  a  = (double*)malloc(NB*NT*sizeof(double));
  pot  = (double*)malloc(NB*NT*sizeof(double));
  // map  = (int*)malloc(M*M*NPC*sizeof(int));
  hipMalloc((void**)&x_dev,  NB * NT * sizeof(double)); // CudaMalloc should be executed once in the host. 
  hipMalloc((void**)&y_dev,  NB * NT * sizeof(double)); 
  hipMalloc((void**)&dx_dev,  NB * NT * sizeof(double)); 
  hipMalloc((void**)&dy_dev,  NB * NT * sizeof(double)); 
  hipMalloc((void**)&vx_dev, NB * NT * sizeof(double)); 
  hipMalloc((void**)&vy_dev, NB * NT * sizeof(double));
  hipMalloc((void**)&fx_dev, NB * NT * sizeof(double)); 
  hipMalloc((void**)&fy_dev, NB * NT * sizeof(double));
  hipMalloc((void**)&a_dev,  NB * NT * sizeof(double)); 
  hipMalloc((void**)&pot_dev,  NB * NT * sizeof(double));
  
 
  hipMalloc((void**)&gate_dev, sizeof(int)); 
  hipMalloc((void**)&list_dev,  NB * NT * NN* sizeof(int)); 
  hipMalloc((void**)&map_dev,  M * M * NPC* sizeof(int)); 
  hipMalloc((void**)&state,  NB * NT * sizeof(hiprandState)); 

  setCurand<<<NB,NT>>>(0, state); // Construction of the cudarand state.  

  init_array_rand<<<NB,NT>>>(x_dev,LB,state);
  init_array_rand<<<NB,NT>>>(y_dev,LB,state);
  init_array<<<NB,NT>>>(a_dev,1.0);
  init_array<<<NB,NT>>>(vx_dev,0.);
  init_array<<<NB,NT>>>(vy_dev,0.);
  init_array<<<NB,NT>>>(pot_dev,0.);
  init_gate_kernel<<<1,1>>>(gate_dev,1);
  init_map_kernel<<<M*M,NPC>>>(map_dev,M);
  cell_map<<<NB,NT>>>(LB,x_dev,y_dev,map_dev,gate_dev,M);
  cell_list<<<NB,NT>>>(LB,x_dev,y_dev,dx_dev,dy_dev,list_dev,map_dev,gate_dev,M);


 
  for(double t=0;t<timemax_bd;t+=dt_bd){
    calc_force_kernel<<<NB,NT>>>(x_dev,y_dev,fx_dev,fy_dev,a_dev,LB,list_dev);
    langevin_kernel<<<NB,NT>>>(x_dev,y_dev,vx_dev,vy_dev,fx_dev,fy_dev,state,noise_intensity,LB);
    init_gate_kernel<<<1,1>>>(gate_dev,0);
    disp_gate_kernel<<<NB,NT>>>(LB,vx_dev,vy_dev,dx_dev,dy_dev,gate_dev,dt_bd);
    init_map_kernel<<<M*M,NPC>>>(map_dev,M);
    cell_map<<<NB,NT>>>(LB,x_dev,y_dev,map_dev,gate_dev,M);
    cell_list<<<NB,NT>>>(LB,x_dev,y_dev,dx_dev,dy_dev,list_dev,map_dev,gate_dev,M);
  } 
  
 
  double amp_vel =sqrt(temp);
  genrand_kernel<<<NB,NT>>>(vx_dev,state,amp_vel);
  genrand_kernel<<<NB,NT>>>(vy_dev,state,amp_vel);
 
 measureTime();  
  for(double t=0;t<timemax_md;t+=dt_md){
    md_kernel_pre<<<NB,NT>>>(x_dev,y_dev,vx_dev,vy_dev,fx_dev,fy_dev,LB);
    calc_force_kernel<<<NB,NT>>>(x_dev,y_dev,fx_dev,fy_dev,a_dev,LB,list_dev);
    md_kernel_post<<<NB,NT>>>(x_dev,y_dev,vx_dev,vy_dev,fx_dev,fy_dev,LB);
    init_gate_kernel<<<1,1>>>(gate_dev,0);
    disp_gate_kernel<<<NB,NT>>>(LB,vx_dev,vy_dev,dx_dev,dy_dev,gate_dev,dt_bd);
    init_map_kernel<<<M*M,NPC>>>(map_dev,M);
    cell_map<<<NB,NT>>>(LB,x_dev,y_dev,map_dev,gate_dev,M);
    cell_list<<<NB,NT>>>(LB,x_dev,y_dev,dx_dev,dy_dev,list_dev,map_dev,gate_dev,M);
  
       if(((int)(t/dt_md))%1000000 == 0){
      calc_energy_kernel<<<NB,NT>>>(x_dev,y_dev,pot_dev,a_dev,LB,list_dev);
      hipMemcpy(pot, pot_dev, NB * NT* sizeof(double),hipMemcpyDeviceToHost);
      hipMemcpy(x, x_dev, NB * NT* sizeof(double),hipMemcpyDeviceToHost);
      hipMemcpy(y, y_dev, NB * NT* sizeof(double),hipMemcpyDeviceToHost);
      hipMemcpy(vx, vx_dev, NB * NT* sizeof(double),hipMemcpyDeviceToHost);
      hipMemcpy(vy, vy_dev, NB * NT* sizeof(double),hipMemcpyDeviceToHost);
      hipMemcpy(a, a_dev, NB * NT* sizeof(double),hipMemcpyDeviceToHost);
      output(x,y,vx,vy,a,pot);
      cout<<"t="<< t <<endl;
    }
    
  }
  sec = measureTime()/1000.;
  cout<<"time(sec):"<<sec<<endl;
 

  hipFree(x_dev);
  hipFree(vx_dev);
  hipFree(y_dev);
  hipFree(vy_dev);
  hipFree(dx_dev);
  hipFree(dy_dev);
  hipFree(pot_dev);
  hipFree(gate_dev);
  hipFree(state);
  free(x); 
  free(vx); 
  free(y); 
  free(vy);
  free(pot); 
  return 0;
}
