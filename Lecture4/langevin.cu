#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#include "../timer.cuh"
#include <math.h>
#include <iostream>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include "MT.h"
using namespace std;
//Using "const", the variable is shared into both gpu and cpu. 
const int  NT = 1024; //Num of the cuda threads.
const int  NP = 1e+6; //Particle number.
const int NB = (NP+NT-1)/NT; //Num of the cuda blocks.
const double dt= 0.01;
const int timemax = 1e+3;
//Langevin parameters
const double zeta = 1.0;
const double temp = 1.0;

//Initiallization of "hiprandState"
__global__ void setCurand(unsigned long long seed, hiprandState *state){
  int i_global = threadIdx.x + blockIdx.x*blockDim.x;
  hiprand_init(seed, i_global, 0, &state[i_global]);
}

//Gaussian random number's generation
__global__ void genrand_kernel(float *result, hiprandState *state){  
  int i_global = threadIdx.x + blockIdx.x*blockDim.x;
  result[i_global] = hiprand_normal(&state[i_global]);
}

//Gaussian random number's generation
__global__ void langevin_kernel(double*x_dev,double *v_dev,hiprandState *state, double noise_intensity){
  int i_global = threadIdx.x + blockIdx.x*blockDim.x;
  if(i_global<NP){
    //  printf("%d,%f\n",i_global,v_dev[i_global]);
    v_dev[i_global] += -v_dev[i_global]*dt+ noise_intensity*hiprand_normal(&state[i_global]);
    x_dev[i_global] += v_dev[i_global]*dt;
  }
}


void init_array(double *x,int Narr, double c){
  for(int i=0;i<Narr;i++) x[i] = c;
}

int main(){
  double *x,*v,*x_dev,*v_dev;
  hiprandState *state; //Cuda state for random numbers
  double sec; //measurred time
  double noise_intensity = sqrt(2.*zeta*temp*dt); //Langevin noise intensity.  
  x = (double*)malloc(NB*NT*sizeof(double));
  v = (double*)malloc(NB*NT*sizeof(double));
  hipMalloc((void**)&x_dev, NB * NT * sizeof(double)); // CudaMalloc should be executed once in the host. 
  hipMalloc((void**)&v_dev, NB * NT * sizeof(double)); 
  hipMalloc((void**)&state, NB * NT * sizeof(hiprandState)); 
  init_array(x,NT*NB,0.);
  init_array(v,NT*NB,0.);
  hipMemcpy(x_dev, x, NB * NT* sizeof(double),hipMemcpyHostToDevice);
  hipMemcpy(v_dev, v, NB * NT* sizeof(double),hipMemcpyHostToDevice);  
  setCurand<<<NB,NT>>>(0, state);
 
  for(double t=0;t<timemax;t+=dt){     
    langevin_kernel<<<NB,NT>>>(x_dev,v_dev,state,noise_intensity);
    //  hipDeviceSynchronize(); // for printf in the device.
  } 
  hipMemcpy(x, x_dev, NB * NT* sizeof(double),hipMemcpyDeviceToHost);
  hipMemcpy(v, v_dev, NB * NT* sizeof(double),hipMemcpyDeviceToHost);
  //  double v_s= sum(v);
  double diffusion=0.0;
  for(int i=0;i<NP;i++)
    diffusion += 0.5*x[i]*x[i]/timemax/NP;
    
  cout <<diffusion<<endl;

  //  cout << sec <<"sec"<<endl;  
  hipFree(x_dev);
  hipFree(v_dev);
  hipFree(state);
  free(x); 
  free(v); 
  return 0;
}
