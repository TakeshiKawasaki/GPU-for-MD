#include <iostream>
#include <hip/hip_runtime.h>
using namespace std;

__global__ void add(int *c, int *a, int *b){
  *c = *a + *b;
}

int main(){
  int a, b, c; //values on the host                                             
  int *a_dev, *b_dev, *c_dev; //values on the device                            

  //Allocate memories on the devices                                            
  hipMalloc((void**)&a_dev, sizeof(int));
  hipMalloc((void**)&b_dev, sizeof(int));
  hipMalloc((void**)&c_dev, sizeof(int));

  //Setup input values                                                          
  a = 2;
  b = 7;

  //Transfer the inputs to device                                               
  hipMemcpy(a_dev, &a, sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(b_dev, &b, sizeof(int), hipMemcpyHostToDevice);

  //Launch add() kernel on the device                                           
  add<<<1, 1>>>(c_dev, a_dev, b_dev);

  //Transfer the output to host                                                 
  hipMemcpy(&c, c_dev, sizeof(int), hipMemcpyDeviceToHost);
  cout << "c: " << c << endl;

  //Free                                                                        
  hipFree(a_dev);
  hipFree(b_dev);
  hipFree(c_dev);
}
