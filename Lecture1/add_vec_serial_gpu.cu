#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include "timer.cuh"

using namespace std;

const int Nv = 1000000;

void setup_vec(int *vec, int a){
  for(int i = 0; i < Nv; i++){
    vec[i] = i * a;
  }
  return;
}


__global__ void add_vec(int *c, int *a, int* b){
  for(int i = 0; i < Nv; i++){
    c[i] = a[i] + b[i];
  }
}

int main(){
  int *a, *b, *c;
  int *a_dev, *b_dev, *c_dev;

  //Allocation
  a = (int*)malloc(Nv * sizeof(int));
  b = (int*)malloc(Nv * sizeof(int));
  c = (int*)malloc(Nv * sizeof(int));

  hipMalloc((void**)&a_dev, Nv * sizeof(int));
  hipMalloc((void**)&b_dev, Nv * sizeof(int));
  hipMalloc((void**)&c_dev, Nv * sizeof(int));

  //Setup input vecs
  setup_vec(a, 1);
  setup_vec(b, 2);

  //Transfer input vecs to device
  hipMemcpy(a_dev, a, Nv * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(b_dev, b, Nv * sizeof(int), hipMemcpyHostToDevice);

  //Launch add_vec() on the device
  double ms;
  measureTime();
  for(int i = 0; i < 1000; i++){
    add_vec<<<1, 1>>>(c_dev, a_dev, b_dev);
  }
  hipMemcpy(c, c_dev, sizeof(int), hipMemcpyDeviceToHost);
  ms = measureTime();
  cout << "Time: " << ms/1000. << "ms" << endl;

  //Free
  free(a);
  free(b);
  free(c);
  hipFree(a_dev);
  hipFree(b_dev);
  hipFree(c_dev);

  return 0;
}
